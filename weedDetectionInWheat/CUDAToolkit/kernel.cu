

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C" __global__ void actualizar(float *posiciones, float *posicionAlfa, float *posicionBeta, float *posicionDelta, float a, int num_pesos, int num_agentes, unsigned long long seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_agentes * num_pesos) {
        hiprandState state;
        hiprand_init(seed, tid, 0, &state);
        
        int n = tid / num_pesos;
        int i = tid % num_pesos;

        float r1 = hiprand_uniform(&state);
        float r2 = hiprand_uniform(&state);
        float A1 = 2 * a * r1 - a;
        float C1 = 2 * r2;

        r1 = hiprand_uniform(&state);
        r2 = hiprand_uniform(&state);
        float A2 = 2 * a * r1 - a;
        float C2 = 2 * r2;

        r1 = hiprand_uniform(&state);
        r2 = hiprand_uniform(&state);
        float A3 = 2 * a * r1 - a;
        float C3 = 2 * r2;

        float posicionAlfa_i = posicionAlfa[i];
        float posicionBeta_i = posicionBeta[i];
        float posicionDelta_i = posicionDelta[i];
        float posicionSolucion_i = posiciones[n * num_pesos + i];

        float distanciaAlfa = fabs(C1 * posicionAlfa_i - posicionSolucion_i);
        float distanciaBeta = fabs(C2 * posicionBeta_i - posicionSolucion_i);
        float distanciaDelta = fabs(C3 * posicionDelta_i - posicionSolucion_i);

        float X1 = posicionAlfa_i - A1 * distanciaAlfa;
        float X2 = posicionBeta_i - A2 * distanciaBeta;
        float X3 = posicionDelta_i - A3 * distanciaDelta;

        posiciones[n * num_pesos + i] = (X1 + X2 + X3) / 3;
    }
}
